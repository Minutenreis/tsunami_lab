#include "hip/hip_runtime.h"
/**
 * @author Justus Dreßler (justus.dressler AT uni-jena.de)
 * @author Thorsten Kröhl (thorsten.kroehl AT uni-jena.de)
 * @author Julius Halank (julius.halank AT uni-jena.de)
 *
 * @section DESCRIPTION
 * Two-dimensional wave propagation patch.
 **/
#include "WavePropagationCUDA.h"
#include "../../solvers/roe/Roe.h"
#include "../../solvers/fWave/FWave.h"

__global__ void setGhostCellsX(tsunami_lab::t_real *io_h, tsunami_lab::t_real *io_hu, tsunami_lab::t_idx i_nx);
__global__ void setGhostCellsY(tsunami_lab::t_real *io_h, tsunami_lab::t_real *io_hv, tsunami_lab::t_idx i_nx, tsunami_lab::t_idx i_ny);
__global__ void initGhostCellsCuda(tsunami_lab::t_real *io_b, tsunami_lab::t_idx i_nx, tsunami_lab::t_idx i_ny);

tsunami_lab::patches::WavePropagationCUDA::WavePropagationCUDA(t_idx i_nCellsx,
                                                               t_idx i_nCellsy,
                                                               bool,
                                                               t_boundary i_boundaryLeft,
                                                               t_boundary i_boundaryRight,
                                                               t_boundary i_boundaryBottom,
                                                               t_boundary i_boundaryTop) : m_nCellsx(i_nCellsx),
                                                                                           m_nCellsy(i_nCellsy),
                                                                                           m_boundaryLeft(i_boundaryLeft),
                                                                                           m_boundaryRight(i_boundaryRight),
                                                                                           m_boundaryBottom(i_boundaryBottom),
                                                                                           m_boundaryTop(i_boundaryTop)
{

    // allocate memory including a single ghost cell on each side (zero initialised)
    t_idx l_size = (m_nCellsx + 2) * (m_nCellsy + 2) * sizeof(t_real);
    hipMallocManaged(&m_h, l_size);
    hipMallocManaged(&m_hu, l_size);
    hipMallocManaged(&m_hv, l_size);
    hipMallocManaged(&m_hTemp, l_size);
    hipMallocManaged(&m_huvTemp, l_size);
    hipMallocManaged(&m_b, l_size);
}

tsunami_lab::patches::WavePropagationCUDA::~WavePropagationCUDA()
{
    hipFree(m_h);
    hipFree(m_hu);
    hipFree(m_hv);
    hipFree(m_hTemp);
    hipFree(m_huvTemp);
    hipFree(m_b);
}

tsunami_lab::t_idx tsunami_lab::patches::WavePropagationCUDA::getCoord(t_idx i_x, t_idx i_y)
{
    return i_x + i_y * (m_nCellsx + 2);
}

void tsunami_lab::patches::WavePropagationCUDA::timeStep(t_real i_scaling)
{
    setGhostCellsX<<<m_nCellsx, m_nCellsy>>>(m_h, m_hu, m_nCellsx);

// init new cell quantities
#pragma omp parallel for simd
    for (t_idx l_cy = 0; l_cy < m_nCellsy + 1; l_cy++)
        for (t_idx l_cx = 0; l_cx < m_nCellsx + 1; l_cx++)
        {
            m_hTemp[getCoord(l_cx, l_cy)] = m_h[getCoord(l_cx, l_cy)];
            m_huvTemp[getCoord(l_cx, l_cy)] = m_hu[getCoord(l_cx, l_cy)];
        }

// iterate over edges and update with Riemann solutions in x direction
#pragma omp parallel for
    for (t_idx l_ey = 0; l_ey < m_nCellsy + 1; l_ey++)
        for (t_idx l_ex = 0; l_ex < m_nCellsx + 1; l_ex++)
        {
            // determine left and right cell-id
            t_idx l_ceL = getCoord(l_ex, l_ey);
            t_idx l_ceR = getCoord(l_ex + 1, l_ey);

            // compute net-updates
            t_real l_netUpdates[2][2];

            solvers::FWave::netUpdates(m_hTemp[l_ceL],
                                       m_hTemp[l_ceR],
                                       m_huvTemp[l_ceL],
                                       m_huvTemp[l_ceR],
                                       m_b[l_ceL],
                                       m_b[l_ceR],
                                       l_netUpdates[0],
                                       l_netUpdates[1]);

            // update the cells' quantities
            m_h[l_ceL] -= i_scaling * l_netUpdates[0][0];
            m_hu[l_ceL] -= i_scaling * l_netUpdates[0][1];

            m_h[l_ceR] -= i_scaling * l_netUpdates[1][0];
            m_hu[l_ceR] -= i_scaling * l_netUpdates[1][1];
        }

    setGhostCellsY<<<m_nCellsx, m_nCellsy>>>(m_h, m_hv, m_nCellsx, m_nCellsy);

// init new cell quantities
#pragma omp parallel for simd
    for (t_idx l_cy = 0; l_cy < m_nCellsy + 1; l_cy++)
        for (t_idx l_cx = 0; l_cx < m_nCellsx + 1; l_cx++)
        {
            m_hTemp[getCoord(l_cx, l_cy)] = m_h[getCoord(l_cx, l_cy)];
            m_huvTemp[getCoord(l_cx, l_cy)] = m_hv[getCoord(l_cx, l_cy)];
        }

// iterate over edges and update with Riemann solutions in y direction
#pragma omp parallel for
    for (t_idx l_ex = 0; l_ex < m_nCellsx + 1; l_ex++)
        for (t_idx l_ey = 0; l_ey < m_nCellsy + 1; l_ey++)
        {
            // determine top and bottom cell-id
            t_idx l_ceB = getCoord(l_ex, l_ey);
            t_idx l_ceT = getCoord(l_ex, l_ey + 1);

            // compute net-updates
            t_real l_netUpdates[2][2];

            solvers::FWave::netUpdates(m_hTemp[l_ceB],
                                       m_hTemp[l_ceT],
                                       m_huvTemp[l_ceB],
                                       m_huvTemp[l_ceT],
                                       m_b[l_ceB],
                                       m_b[l_ceT],
                                       l_netUpdates[0],
                                       l_netUpdates[1]);

            // update the cells' quantities
            m_h[l_ceB] -= i_scaling * l_netUpdates[0][0];
            m_hv[l_ceB] -= i_scaling * l_netUpdates[0][1];

            m_h[l_ceT] -= i_scaling * l_netUpdates[1][0];
            m_hv[l_ceT] -= i_scaling * l_netUpdates[1][1];
        }
}

// __global__ void tsunami_lab::patches::WavePropagationCUDA::setGhostCellsX(tsunami_lab::t_real *io_h, tsunami_lab::t_real *io_hu, tsunami_lab::t_idx i_nx)
__global__ void setGhostCellsX(tsunami_lab::t_real *io_h, tsunami_lab::t_real *io_hu, tsunami_lab::t_idx i_nx)
{
    tsunami_lab::t_idx l_x = blockIdx.x * blockDim.x + threadIdx.x;
    tsunami_lab::t_idx l_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (l_x == 0)
    {
        io_h[i_nx * l_y] = io_h[1 + i_nx * l_y];
        io_hu[i_nx * l_y] = io_hu[1 + i_nx * l_y];
    }
    else if (l_x == i_nx - 1)
    {
        io_h[l_x + i_nx * l_y] = io_h[l_x - 1 + i_nx * l_y];
        io_hu[l_x + i_nx * l_y] = io_hu[l_x - 1 + i_nx * l_y];
    }
}

// __global__ void tsunami_lab::patches::WavePropagationCUDA::setGhostCellsY(tsunami_lab::t_real *io_h, tsunami_lab::t_real *io_hv, tsunami_lab::t_idx i_nx, tsunami_lab::t_idx i_ny)
__global__ void setGhostCellsY(tsunami_lab::t_real *io_h, tsunami_lab::t_real *io_hv, tsunami_lab::t_idx i_nx, tsunami_lab::t_idx i_ny)
{
    tsunami_lab::t_idx l_x = blockIdx.x * blockDim.x + threadIdx.x;
    tsunami_lab::t_idx l_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (l_y == 0)
    {
        io_h[l_x] = io_h[l_x + i_nx];
        io_hv[l_x] = io_hv[l_x + i_nx];
    }
    else if (l_y == i_ny - 1)
    {
        io_h[l_x + i_nx * l_y] = io_h[l_x + i_nx * (l_y - 1)];
        io_hv[l_x + i_nx * l_y] = io_hv[l_x + i_nx * (l_y - 1)];
    }
}

void tsunami_lab::patches::WavePropagationCUDA::initGhostCells()
{
    initGhostCellsCuda<<<m_nCellsx, m_nCellsy>>>(m_b, m_nCellsx, m_nCellsy);
}

// __global__ void tsunami_lab::patches::WavePropagationCUDA::initGhostCellsCuda(tsunami_lab::t_real *io_b, tsunami_lab::t_idx i_nx, tsunami_lab::t_idx i_ny)
__global__ void initGhostCellsCuda(tsunami_lab::t_real *io_b, tsunami_lab::t_idx i_nx, tsunami_lab::t_idx i_ny)
{
    tsunami_lab::t_idx l_x = blockIdx.x * blockDim.x + threadIdx.x;
    tsunami_lab::t_idx l_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (l_x == 0)
    {
        io_b[l_x + blockDim.x * l_y] = io_b[l_x + 1 + blockDim.x * l_y];
    }
    else if (l_x == blockDim.x - 1)
    {
        io_b[l_x + blockDim.x * l_y] = io_b[l_x - 1 + blockDim.x * l_y];
    }
    else if (l_y == 0)
    {
        io_b[l_x] = io_b[l_x + i_nx];
    }
    else if (l_y == i_ny - 1)
    {
        io_b[l_x + i_nx * l_y] = io_b[l_x + i_nx * (l_y - 1)];
    }
}